#ifndef SIZE
#define SIZE 32
#endif

#ifndef REPETITIONS
#define REPETITIONS 5
#endif

#include <iostream>
#include <vector>
#include <unordered_map>
#include "FFT.h"
#include "ChunkTesting.cuh"
#include "LinearizedPoly.h"
#include "Basis.h"
#include "F2XE.h"
#include "Chunk.h"

int main(int argc, char* argv[])
{
	std::vector<int> a;
	std::unordered_map<int, int> b;

}
